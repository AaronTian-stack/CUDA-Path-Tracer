#include "hip/hip_runtime.h"
#include "cuda_test.h"

#include <cstdio>

void kernelWrapper()
{
    helloKernel<<<1, 1>>>();
	hipDeviceSynchronize();
}

__global__ void helloKernel() 
{
    printf("device\n");
}