#include "hip/hip_runtime.h"
#include "intersection.h"

#include "util.h"

#define multiply_mv(m, v) glm::vec3((m) * (v))

__host__ __device__ float box_intersection_test(
    Geom box,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    Ray q;
    q.origin = multiply_mv(box.inverseTransform, glm::vec4(r.origin, 1.0f));
    q.direction = glm::normalize(multiply_mv(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float tmin = -1e38f;
    float tmax = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? 1.f : -1.f;
            if (ta > 0 && ta > tmin)
            {
                tmin = ta;
                tmin_n = n;
            }
            if (tb < tmax)
            {
                tmax = tb;
                tmax_n = n;
            }
        }
    }

    if (tmax >= tmin && tmax > 0.f)
    {
        outside = true;
        if (tmin <= 0)
        {
            tmin = tmax;
            tmin_n = tmax_n;
            outside = false;
        }
        intersectionPoint = multiply_mv(box.transform, glm::vec4(get_point_on_ray(q, tmin), 1.0f));
        normal = glm::normalize(multiply_mv(box.invTranspose, glm::vec4(tmin_n, 0.0f)));

  //      auto t = glm::length(r.origin - intersectionPoint);;
		//printf("cube fuck t: %f\n", t);

        return glm::length(r.origin - intersectionPoint);
    }

    return -1.f;
}

__host__ __device__ float sphere_intersection_test(
    Geom sphere,
    Ray r,
    glm::vec3& intersectionPoint,
    glm::vec3& normal,
    bool& outside)
{
    float radius = 0.5f;

    glm::vec3 ro = multiply_mv(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiply_mv(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float vDotDirection = glm::dot(rt.origin, rt.direction);
    float radicand = vDotDirection * vDotDirection - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0.f)
    {
        return -1;
    }

    float squareRoot = sqrt(radicand);
    float firstTerm = -vDotDirection;
    float t1 = firstTerm + squareRoot;
    float t2 = firstTerm - squareRoot;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    else if (t1 > 0 && t2 > 0)
    {
        t = min(t1, t2);
        outside = true;
    }
    else
    {
        t = max(t1, t2);
        outside = false;
    }

    glm::vec3 objspaceIntersection = get_point_on_ray(rt, t);

    intersectionPoint = multiply_mv(sphere.transform, glm::vec4(objspaceIntersection, 1.f));
    normal = glm::normalize(multiply_mv(sphere.invTranspose, glm::vec4(objspaceIntersection, 0.f)));

    return glm::length(r.origin - intersectionPoint);
}
