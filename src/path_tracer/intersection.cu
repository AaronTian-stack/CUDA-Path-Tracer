#include "hip/hip_runtime.h"
#include "intersection.h"

#include "util.h"

#define multiply_mv(m, v) glm::vec3((m) * (v))

__host__ __device__ float box_intersection_test(
    Geom box,
    Ray r,
    glm::vec3& intersection_point,
    glm::vec3& normal,
    bool& outside)
{
    Ray q;
    q.origin = multiply_mv(box.inverseTransform, glm::vec4(r.origin, 1.0f));
    q.direction = glm::normalize(multiply_mv(box.inverseTransform, glm::vec4(r.direction, 0.0f)));

    float t_min = -1e38f;
    float t_max = 1e38f;
    glm::vec3 tmin_n;
    glm::vec3 tmax_n;
    for (int xyz = 0; xyz < 3; ++xyz)
    {
        float qdxyz = q.direction[xyz];
        /*if (glm::abs(qdxyz) > 0.00001f)*/
        {
            float t1 = (-0.5f - q.origin[xyz]) / qdxyz;
            float t2 = (0.5f - q.origin[xyz]) / qdxyz;
            float ta = glm::min(t1, t2);
            float tb = glm::max(t1, t2);
            glm::vec3 n;
            n[xyz] = t2 < t1 ? 1.0f : -1.0f;
            if (ta > 0 && ta > t_min)
            {
                t_min = ta;
                tmin_n = n;
            }
            if (tb < t_max)
            {
                t_max = tb;
                tmax_n = n;
            }
        }
    }

    if (t_max >= t_min && t_max > 0.0f)
    {
        outside = true;
        if (t_min <= 0)
        {
            t_min = t_max;
            tmin_n = tmax_n;
            outside = false;
        }
        intersection_point = multiply_mv(box.transform, glm::vec4(get_point_on_ray(q, t_min), 1.0f));
        normal = glm::normalize(multiply_mv(box.invTranspose, glm::vec4(tmin_n, 0.0f)));

        return glm::length(r.origin - intersection_point);
    }

    return -1.0f;
}

__host__ __device__ float sphere_intersection_test(
    Geom sphere,
    Ray r,
    glm::vec3& intersection_point,
    glm::vec3& normal,
    bool& outside)
{
    float radius = 0.5f;

    glm::vec3 ro = multiply_mv(sphere.inverseTransform, glm::vec4(r.origin, 1.0f));
    glm::vec3 rd = glm::normalize(multiply_mv(sphere.inverseTransform, glm::vec4(r.direction, 0.0f)));

    Ray rt;
    rt.origin = ro;
    rt.direction = rd;

    float v_dot_direction = glm::dot(rt.origin, rt.direction);
    float radicand = v_dot_direction * v_dot_direction - (glm::dot(rt.origin, rt.origin) - powf(radius, 2));
    if (radicand < 0.0f)
    {
        return -1;
    }

    float square_root = sqrt(radicand);
    float first_term = -v_dot_direction;
    float t1 = first_term + square_root;
    float t2 = first_term - square_root;

    float t = 0;
    if (t1 < 0 && t2 < 0)
    {
        return -1;
    }
    if (t1 > 0 && t2 > 0)
    {
	    t = min(t1, t2);
	    outside = true;
    }
    else
    {
	    t = max(t1, t2);
	    outside = false;
    }

    glm::vec3 objspace_intersection = get_point_on_ray(rt, t);

    intersection_point = multiply_mv(sphere.transform, glm::vec4(objspace_intersection, 1.0f));
    normal = glm::normalize(multiply_mv(sphere.invTranspose, glm::vec4(objspace_intersection, 0.0f)));

    return glm::length(r.origin - intersection_point);
}
